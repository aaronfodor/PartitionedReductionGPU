#include "hip/hip_runtime.h"
#include "cuda_functions.h"

// number of threads in a block
const int BLOCK_SIZE = 32;

__global__ void partitioned_reduce_kernel(float* device_input){
    int idx = threadIdx.x;
    int currentIdx = blockIdx.x * blockDim.x + threadIdx.x;

    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1){
        if(idx < s){
            device_input[currentIdx] = max(device_input[currentIdx], device_input[currentIdx + s]);
        }
        __syncthreads();
    }

    if(threadIdx.x == 0){
        device_input[blockIdx.x] = device_input[currentIdx];
    }
}

float CUDA_FUNCTIONS::partitioned_reduction(float* input_array, const size_t dataSize){
    float* device_input = nullptr;
    float result;

    int numBlocks = ceil(dataSize / BLOCK_SIZE) + 1;

    dim3 dimGrid = numBlocks;
    dim3 dimBlock = BLOCK_SIZE;
    int numMergeIterations = ceil((log(numBlocks) / log(BLOCK_SIZE)) + 1);

    hipMalloc(&device_input, sizeof(int) * dataSize);
    hipMemcpy(device_input, input_array, sizeof(int) * dataSize, hipMemcpyHostToDevice);

    for(unsigned int i = 0; i < numMergeIterations; i++){
        partitioned_reduce_kernel<<<dimGrid, dimBlock>>>(device_input);
    }

    hipMemcpy(&result, device_input, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(device_input);

    return result;
}
